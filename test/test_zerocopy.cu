#include <catch2/catch.hpp>

#include "cmm/cmm.hpp"
#include "writer.hpp"

TEMPLATE_TEST_CASE("zerocopy", "[cuda][template]", int, double) {
    typedef cmm::ZeroCopy<TestType> Allocator;

    SECTION("ctor") {
        Allocator a;
        TestType *p = a.allocate(10);
        writer<<<10, 10>>>(p, 10);
        hipDeviceSynchronize();
        a.deallocate(p, 10);
    }
}